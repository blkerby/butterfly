#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include <stdio.h>
#include <time.h>
#include <math.h>


#define gpuCheck(f) { gpuCheckFunc((f), __FILE__, __LINE__); }

inline void gpuCheckFunc(hipError_t err, const char *file, int line){
   if (err != hipSuccess) {
      fprintf(stderr, "CUDA error (%s:%d): %s\n", file, line, hipGetErrorString(err));
      exit(1);
   }
}

template <typename scalar_t>
__device__ scalar_t reduce_add_block(scalar_t x, scalar_t *s_tmp) {
    // First reduce across the warp:
    for (int m = 1; m < 32; m <<= 1){
        x += __shfl_xor_sync(0xffffffff, x, m);
    }

    // Now reduce across the block
    if (threadIdx.x % 32 == 0) {
        s_tmp[threadIdx.x / 32] = x;
    }
    __syncthreads();
    if (threadIdx.x < blockDim.x / 32) {
        x = s_tmp[threadIdx.x];
    } else {
        x = 0.0;
    }
    for (int m = 1; m < blockDim.x / 32; m <<= 1){
        x += __shfl_xor_sync(0xffffffff, x, m);
    }
    return x;
}

__device__ void reduce_add_global(float x, float *s_tmp, float *g_out) {
    x = reduce_add_block(x, s_tmp);
    if (threadIdx.x == 0) {
        atomicAdd(g_out, x);
    }
}

#if __CUDA_ARCH__ >= 600
__device__ void reduce_add_global(double x, double *s_tmp, double *g_out) {
    x = reduce_add_block(x, s_tmp);
    if (threadIdx.x == 0) {
        atomicAdd(g_out, x);
    }
}
#else
__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ void reduce_add_global(double x, double *s_tmp, double *g_out) {
    x = reduce_add_block(x, s_tmp);
    if (threadIdx.x == 0) {
        atomicAddDouble(g_out, x);
    }
}
#endif


// Saturates the GPU global memory bandwidth with very low utilization of the ALUs. 
// To get better efficiency we would need to fuse multiple layers together to 
// reduce the amount of GPU memory loads and stores.
template <typename scalar_t>
__global__ void cuda_butterfly_forward_slow_kernel(
    const scalar_t *data_in,
    const scalar_t *angles,
    scalar_t *data_out,
    int data_stride,
    int half_width
) {
    // Load the angle for this thread's switch, and compute the corresponding weights.
    scalar_t angle = angles[blockIdx.y];
    scalar_t a = cos(angle);
    scalar_t b = sin(angle);
    
    // Load the input data from GPU global memory
    int data_idx_in = 2 * blockIdx.y * data_stride + threadIdx.x + blockDim.x * blockIdx.x;
    scalar_t x0 = data_in[data_idx_in];
    scalar_t y0 = data_in[data_idx_in + data_stride];

    // Compute the output data
    scalar_t x1 = a * x0 + b * y0;
    scalar_t y1 = -b * x0 + a * y0;

    // Write the output data to GPU global memory
    int data_idx_out = blockIdx.y * data_stride + threadIdx.x + blockDim.x * blockIdx.x;
    data_out[data_idx_out] = x1;
    data_out[data_idx_out + data_stride * half_width] = y1;
}

template <typename scalar_t>
__global__ void cuda_butterfly_backward_slow_kernel(
    const scalar_t *data_in,
    const scalar_t *angles,
    const scalar_t *grad_in,
    scalar_t *grad_out,
    scalar_t *grad_angles_accum,
    int data_stride,
    int half_width
) {
    // Load the angle for this thread's switch, and compute the corresponding weights.
    scalar_t angle = angles[blockIdx.y];
    scalar_t a = cos(angle);
    scalar_t b = sin(angle);
    
    // Load the input gradient
    int data_idx_in = blockIdx.y * data_stride + threadIdx.x + blockDim.x * blockIdx.x;
    scalar_t dx1 = grad_in[data_idx_in];
    scalar_t dy1 = grad_in[data_idx_in + data_stride * half_width];

    // Compute the output gradient for continuing backpropagation into earlier layers
    scalar_t dx0 = a * dx1 - b * dy1;
    scalar_t dy0 = b * dx1 + a * dy1;

    // Write the output gradient to GPU global memory
    int data_idx_out = 2 * blockIdx.y * data_stride + threadIdx.x + blockDim.x * blockIdx.x;
    grad_out[data_idx_out] = dx0;
    grad_out[data_idx_out + data_stride] = dy0;

    // Accumulate the gradient for the angles in the current layer
    __shared__ scalar_t tmp[32];
    scalar_t x1 = data_in[data_idx_in];
    scalar_t y1 = data_in[data_idx_in + data_stride * half_width];
    scalar_t g = y1*dx1 - x1*dy1;
    reduce_add_global(g, tmp, &grad_angles_accum[blockIdx.y]);
}

void cuda_butterfly_forward_slow(at::Tensor data_in, at::Tensor angles, at::Tensor data_out) {
    int dimBlock = 256;
    dim3 dimGrid(data_in.size(1) / dimBlock, angles.size(0));

    AT_DISPATCH_FLOATING_TYPES(data_in.type(), "test_cuda_double", ([&] {
        cuda_butterfly_forward_slow_kernel<scalar_t><<<dimGrid, dimBlock>>>(
            data_in.data<scalar_t>(),
            angles.data<scalar_t>(),
            data_out.data<scalar_t>(),
            data_in.size(1),
            data_in.size(0) / 2
        );
        gpuCheck( hipGetLastError() )
    }));
}


void cuda_butterfly_backward_slow(
    at::Tensor data_in,
    at::Tensor angles, 
    at::Tensor grad_in,
    at::Tensor grad_out,
    at::Tensor grad_angles_accum
) {
    int dimBlock = 256;
    dim3 dimGrid(data_in.size(1) / dimBlock, angles.size(0));

    AT_DISPATCH_FLOATING_TYPES(data_in.type(), "test_cuda_double", ([&] {
        cuda_butterfly_backward_slow_kernel<scalar_t><<<dimGrid, dimBlock>>>(
            data_in.data<scalar_t>(),
            angles.data<scalar_t>(),
            grad_in.data<scalar_t>(),
            grad_out.data<scalar_t>(),
            grad_angles_accum.data<scalar_t>(),
            data_in.size(1),
            data_in.size(0) / 2
        );
        gpuCheck( hipGetLastError() )
    }));
}
